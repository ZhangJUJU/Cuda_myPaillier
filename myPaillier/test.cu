#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include "mypail.cuh"

pubkey pub_d;
prvkey prv_d;

__global__ void test(pubkey* pub, prvkey* prv) {
	printf("printing device memory: \n");
	
	setup(pub, prv);
	printf("\n");
}

int main() {
	printf("program starting... \n");

	pubkey pub;
	prvkey prv;
	
	initial(&pub, &prv);
	initial(&pub_d, &prv_d);

	unsigned x;
	unsigned* px = &x;
	size_t size = sizeof(unsigned);

	hipMalloc(&pub_d.n,size);
	hipMalloc(&pub_d.g, size);
	hipMalloc(&prv_d.lamda, size);
	hipMalloc(&prv_d.mu, size);
	/*
	unsigned dn, dg, dlamda, dmu = 0;
	
	hipMemset(pub_d.n, dn, size);
	hipMemset(pub_d.g, dg, size);
	hipMemset(prv_d.lamda, dlamda, size);
	hipMemset(prv_d.mu, dmu, size);
	*/
	test <<<1, 1 >>> (&pub_d, &prv_d);

	hipDeviceSynchronize();

	hipMemcpy(px, pub_d.n, size, hipMemcpyDeviceToHost);
	hipMemcpy(pub.n, pub_d.n, size, hipMemcpyDeviceToHost);
	hipMemcpy(pub.g, pub_d.g, size, hipMemcpyDeviceToHost);
	hipMemcpy(prv.lamda, prv_d.lamda, size, hipMemcpyDeviceToHost);
	hipMemcpy(prv.mu,prv_d.mu, size, hipMemcpyDeviceToHost);

	//hipMemcpy(prv, prv_d, sizeof(prvkey), hipMemcpyDeviceToHost);

	

	hipFree(pub_d.n);
	hipFree(pub_d.g);
	hipFree(prv_d.lamda);
	hipFree(prv_d.mu);
	
	printf("printing host memory: \n");

	printf("n: %d\n", *pub.n);
	printf("g: %d\n", *pub.g);
	printf("lamda: %d\n", *prv.lamda);
	printf("mu: %d\n", *prv.mu);

	printf("x: %d\n", *px);


	printf("program ending... \n");
}