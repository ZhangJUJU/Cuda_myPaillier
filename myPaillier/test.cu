#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "" //__global__
#include "stdio.h"
#include "mypail.cuh"
#include <time.h> //time()



__global__ void test(pubkey pub, prvkey prv) {
	printf("printing device memory (pub_d, prv_d): \n\n");

	//setting up device variable (generating public key and device key)
	setup(pub, prv);
	
	printf("\n");
}

int main() {
	printf("\nprogram starting... \n\n");

	//public key and private key for host
	pubkey pub;
	prvkey prv;
	//public key and private key for device
	pubkey pub_d;
	prvkey prv_d;
	//defining size
	size_t size = sizeof(unsigned);  
	//allocating device memory for device variables
	hipMalloc(&pub_d.n,size);
	hipMalloc(&pub_d.g, size);
	hipMalloc(&prv_d.lamda, size);
	hipMalloc(&prv_d.mu, size);
	//running function on device
	test <<<1, 1 >>> (pub_d,prv_d);
	//waiting device to finish its job
	hipDeviceSynchronize();
	//copy device memory to host memory
	hipMemcpy(pub.n, pub_d.n, size, hipMemcpyDeviceToHost);
	hipMemcpy(pub.g, pub_d.g, size, hipMemcpyDeviceToHost);
	hipMemcpy(prv.lamda, prv_d.lamda, size, hipMemcpyDeviceToHost);
	hipMemcpy(prv.mu,prv_d.mu, size, hipMemcpyDeviceToHost);
	//free the device memory
	hipFree(pub_d.n);
	hipFree(pub_d.g);
	hipFree(prv_d.lamda);
	hipFree(prv_d.mu);
	//printing the host result
	printf("printing host memory (pub, prv): \n\n");

	printf("n: %d\n", *pub.n);
	printf("g: %d\n", *pub.g);
	printf("lamda: %d\n", *prv.lamda);
	printf("mu: %d\n", *prv.mu);


	printf("\nprogram ending... \n\n");
}